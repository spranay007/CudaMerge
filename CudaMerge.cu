#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <vector>
#include <cstdlib>
#include <chrono>
#include <algorithm>
#include <random>

#define THREADS_PER_BLOCK 512

// Data[], size, threads
void mergesort(long*, long, dim3, dim3);
// A[], B[], size, width, slices
__global__ void gpu_mergesort(long*, long*, long, long, long);
__device__ void gpu_bottomUpMerge(long*, long*, long, long, long);
void cpu_mergesort(long*, long);
void merge(long*, long*, long*, long, long);
bool compareArrays(long* arr1, long* arr2, long size);

// Profiling
long long tm();

#define min(a, b) (a < b ? a : b)

// Generate random numbers
void generateRandomNumbers(std::vector<long>& numbers, long size) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<long> dis(1, 1000);

    for (long i = 0; i < size; ++i) {
        numbers.push_back(dis(gen));
    }
}

void printHelp(char* program) {
    std::cout << "usage: " << program << " <number_of_elements>\n";
}

int main(int argc, char** argv) {
    // Check if the number of elements is provided
    if (argc < 2) {
        std::cout << "Number of elements not provided.\n";
        printHelp(argv[0]);
        return -1;
    }

    long size = std::atol(argv[1]);
    if (size <= 0) {
        std::cout << "Invalid number of elements.\n";
        return -1;
    }

    dim3 threadsPerBlock(THREADS_PER_BLOCK, 1, 1);
    dim3 blocksPerGrid((size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1, 1);

    // Generate random numbers
    std::vector<long> numbers;
    generateRandomNumbers(numbers, size);

    long* cpu_result = new long[size];
    long* gpu_result = new long[size];

    // GPU Sorting
    auto start_gpu = std::chrono::steady_clock::now();
    // Perform GPU mergesort
    mergesort(numbers.data(), size, threadsPerBlock, blocksPerGrid);
    auto end_gpu = std::chrono::steady_clock::now();
    auto gpu_sort_time = std::chrono::duration_cast<std::chrono::microseconds>(end_gpu - start_gpu).count();

    // CPU Sorting
    auto start_cpu = std::chrono::steady_clock::now();
    // Perform CPU mergesort
    cpu_mergesort(numbers.data(), size);
    auto end_cpu = std::chrono::steady_clock::now();
    auto cpu_sort_time = std::chrono::duration_cast<std::chrono::microseconds>(end_cpu - start_cpu).count();

    std::cout << "GPU Sorting Time: " << gpu_sort_time << " microseconds\n";
    std::cout << "CPU Sorting Time: " << cpu_sort_time << " microseconds\n";

    hipMemcpy(gpu_result, numbers.data(), size * sizeof(long), hipMemcpyHostToDevice);

    if (compareArrays(cpu_result, gpu_result, size)) {
        std::cout << "Results matched\n";
    }
    else {
        std::cout << "Results did not match\n";
    }

    delete[] cpu_result;
    delete[] gpu_result;

    return 0;
}

void mergesort(long* data, long size, dim3 threadsPerBlock, dim3 blocksPerGrid) {
    
    //The GPU mergesort algorithm swaps data between D_data and D_swp as needed during its execution.

    long* D_data;   // input data to be sorted (from the host) is copied, used as the source array for the GPU mergesort algorithm.
    long* D_swp;    // array in device memory which serves as temporary storage during the sorting process
    
    long width_temp = 2;
    long slices_temp = (size) / (width_temp * THREADS_PER_BLOCK);

    checkCudaErrors(hipMalloc((void**)&D_data, size * sizeof(long)));
    checkCudaErrors(hipMalloc((void**)&D_swp, size * sizeof(long)));
    // Data from the input array is is coppied from host memory to device memory
    checkCudaErrors(hipMemcpy(D_data, data, size * sizeof(long), hipMemcpyHostToDevice));

    gpu_mergesort << <blocksPerGrid, threadsPerBlock, size * sizeof(long) >> > (D_data, D_swp, size, width_temp, slices_temp);

    checkCudaErrors(hipMemcpy(data, D_data, size * sizeof(long), hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(D_data));
    checkCudaErrors(hipFree(D_swp));
}

__global__ void gpu_mergesort(long* source, long* dest, long size, long width, long slices) {
    extern __shared__ long shared_data[];

    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    long start = width * idx * slices,
        middle,
        end;

    for (long slice = 0; slice < slices; slice++) {
        if (start >= size)
            break;

        // Load data into shared memory
        long sharedIdx = threadIdx.x;
        while (sharedIdx < min(start + width, size)) {
            shared_data[sharedIdx] = source[sharedIdx];
            sharedIdx += blockDim.x;
        }
        __syncthreads();

        middle = min(start + (width >> 1), size);
        end = min(start + width, size);
        gpu_bottomUpMerge(shared_data, dest, 0, middle - start, end - start);
        __syncthreads();

        // Write back the sorted data from shared memory to global memory
        sharedIdx = threadIdx.x;
        while (sharedIdx < end - start) {
            dest[start + sharedIdx] = shared_data[sharedIdx];
            sharedIdx += blockDim.x;
        }
        __syncthreads();

        start += width;
    }
}

__device__ void gpu_bottomUpMerge(long* source, long* dest, long start, long middle, long end) {
    long i = start;
    long j = middle;
    for (long k = start; k < end; k++) {
        if (i < middle && (j >= end || source[i] < source[j])) {
            dest[k] = source[i];
            i++;
        }
        else {
            dest[k] = source[j];
            j++;
        }
    }
}

void cpu_mergesort(long* data, long size) {
    if (size <= 1)
        return;

    long mid = size / 2;

    cpu_mergesort(data, mid);
    cpu_mergesort(data + mid, size - mid);

    merge(data, data, data + mid, mid, size - mid);
}

void merge(long* result, long* left, long* right, long size_left, long size_right) {
    long i = 0, j = 0, k = 0;
    std::vector<long> merged(size_left + size_right);

    while (i < size_left && j < size_right) {
        if (left[i] <= right[j])
            merged[k++] = left[i++];
        else
            merged[k++] = right[j++];
    }

    while (i < size_left)
        merged[k++] = left[i++];

    while (j < size_right)
        merged[k++] = right[j++];

    std::copy(merged.begin(), merged.end(), result);
}

bool compareArrays(long* arr1, long* arr2, long size) {
    for (long i = 0; i < size; i++) {
        if (arr1[i] != arr2[i]) {
            return false;
        }
    }
    return true;
}
